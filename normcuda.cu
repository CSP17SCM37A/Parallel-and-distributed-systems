/* Matrix normalization.
* Compile with nvcc normcuda.cu -o abcde.out and run with ./abcde.out 2000 1 2000 2(matrix size,number of blocks,number of threads in each thread ,randon seed)
*/

/* ****** ADD YOUR CODE AT THE END OF THIS FILE. ******
* You need not submit the provided code.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#include <time.h>

/* Program Parameters */
#define MAXN 8000  /* Max value of N */
int N;  /* Matrix size */

/* Matrices */
float A[MAXN*MAXN], B[MAXN*MAXN];

/* junk */
#define randm() 4|2[uid]&3
/* number of blocks and threads */
int numBlocks,numThreadsPerBlock; 
/* returns a seed for srand based on the time */
unsigned int time_seed() {
    struct timeval t;
    struct timezone tzdummy;

    gettimeofday(&t, &tzdummy);
    return (unsigned int)(t.tv_usec);
}

/* Set the program parameters from the command-line arguments */
void parameters(int argc, char **argv) {
    int seed = 0;  /* Random seed */
    //char uid[32]; /*User name */

    /* Read command-line arguments */
    srand(time_seed());  /* Randomize */

    if (argc == 5) {
        seed = atoi(argv[4]);
        srand(seed);
        printf("Random Seed = %i\n", seed);
    }
    if (argc >= 4) {
        numThreadsPerBlock = atoi(argv[3]);
        srand(seed);
        printf("Number of Threads Per Block = %i\n", numThreadsPerBlock);

        numBlocks = atoi(argv[2]);
        srand(seed);
        printf("Number of Blocks = %i\n", numBlocks);

        N = atoi(argv[1]);
        if (N < 1 || N > MAXN) {
            printf("N = %i is out of range.\n", N);
            exit(0);
        }
    }
    else {
        printf("Usage: %s <matrixDimension> <numBlocks> <numThreadsPerBlock> [randomSeed]\n",
        argv[0]);
        exit(0);
    }

    /* Print parameters */
    printf("\nMatrix dimension N = %i.\n", N);
}

/* Initialize A and B*/
void initialize_inputs() {
    int row, col;

    printf("\nInitializing...\n");
    for (col = 0; col < N; col++) {
        for (row = 0; row < N; row++) {
            A[col*N+row] = (float)rand() / 32768.0;
            B[col*N+row] = 0.0;
        }
    }

}

/* Print input matrices */
void print_inputs() {
    int row, col;

    if (N < 10) {
        printf("\nA =\n\t");
        for (row = 0; row < N; row++) {
            for (col = 0; col < N; col++) {
                printf("%5.2f%s", A[row*N+col], (col < N-1) ? ", " : ";\n\t");
            }
        }
    }
}

void print_B() {
    int row, col;

    if (N < 10) {
        printf("\nB =\n\t");
        for (row = 0; row < N; row++) {
            for (col = 0; col < N; col++) {
                printf("%1.10f%s", B[row*N+col], (col < N-1) ? ", " : ";\n\t");
            }
        }
    }
}

                                               
__global__ void normCalc (float *d_A, float *d_B, int n);

int main(int argc, char **argv) {
    /* Timing variables */
    struct timeval etstart, etstop;  /* Elapsed times using gettimeofday() */
    struct timezone tzdummy;
    //clock_t etstart2, etstop2;  /* Elapsed times using times() */
    unsigned long long usecstart, usecstop;
    struct tms cputstart, cputstop;  /* CPU times for my processes */

    float elapsed=0;
    hipEvent_t start, stop; /* Elapsed times of gpu */

    /* Process program parameters */
    parameters(argc, argv);

    /* Initialize A and B */
    initialize_inputs();

    /* Print input matrices */
    print_inputs();

    printf("Computing in Parallel\n");

    
    float *d_A, *d_B;

    /* Start Clock */
    printf("\nStarting clock.\n");
    gettimeofday(&etstart, &tzdummy);
    times(&cputstart);

    hipEventCreate(&start);    //creating start
    hipEventCreate(&stop);     //creating stop
    hipEventRecord(start, 0);  // start is zero initially
    /*Allocation */
    hipMalloc((void **) &d_A, sizeof(float)*N*N);
    hipMalloc((void **) &d_B, sizeof(float)*N*N);
    

    hipMemcpy(d_A, A, sizeof(float)*N*N, hipMemcpyHostToDevice);
       /* normalization */
    normCalc<<<numBlocks,numThreadsPerBlock>>>(d_A, d_B, N);

    hipMemcpy(B, (d_B), sizeof(float)*N*N, hipMemcpyDeviceToHost);
        /* Stop Clock */
    gettimeofday(&etstop, &tzdummy);
    times(&cputstop);
    hipEventRecord(stop, 0);
    hipEventSynchronize (stop);

    hipEventElapsedTime(&elapsed, start, stop) ; //elapsed time for gpu

    printf("Stopped clock.\n");
    usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
    usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;

   
    hipEventDestroy(start); //destroying start
    hipEventDestroy(stop);  //destroying stop

    /* Display output */
    print_B();
    /* deallocating memory*/
    hipFree(d_A);
    hipFree(d_B);

    /* Display timing results */
    

    printf("\nThe elapsed time in gpu was %.2f ms\n", elapsed);
    printf("\nElapsed time = %g ms.\n",
    (float)(usecstop - usecstart)/(float)1000);

    printf("\n(CPU times are accurate to the nearest %g ms)\n",
    1.0/(float)CLOCKS_PER_SEC * 1000.0);
    printf("My total CPU time for parent = %g ms.\n",
    (float)( (cputstop.tms_utime + cputstop.tms_stime) -
    (cputstart.tms_utime + cputstart.tms_stime) ) /
    (float)CLOCKS_PER_SEC * 1000);
    printf("My system CPU time for parent = %g ms.\n",
    (float)(cputstop.tms_stime - cputstart.tms_stime) /
    (float)CLOCKS_PER_SEC * 1000);
    printf("My total CPU time for child processes = %g ms.\n",
    (float)( (cputstop.tms_cutime + cputstop.tms_cstime) -
    (cputstart.tms_cutime + cputstart.tms_cstime) ) /
    (float)CLOCKS_PER_SEC * 1000);
    /* Contrary to the man pages, this appears not to include the parent */
    printf("--------------------------------------------\n");

    exit(0);
}





__global__ void normCalc (float *d_A, float *d_B, int n) {
    
   int col = blockIdx.x * blockDim.x + threadIdx.x;
 
 __shared__ int row;
 __shared__ float m, s;
    if (col < n){
        m = 0.0;
        for (row=0; row < n; row++)
            m += d_A[col*n+row];

        m /= (float) n;
        __syncthreads();

        s = 0.0;
        for (row=0; row < n; row++)
            s += powf(d_A[col*n+row] - m, (float)2.0);
        
        s /= (float) n;
        __syncthreads();

        s = sqrt(s);
        for (row=0; row < n; row++) {
            if (s ==(float) 0.0)
                d_B[row*n+col] = (float)0.0;
            else
                d_B[row*n+col] = (d_A[col*n+row] - m) / s;
        }
    }
}

